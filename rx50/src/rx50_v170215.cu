#include "hip/hip_runtime.h"
/*
* The software-defined radio receiver V170215
* The authors would thank NICT for original source codes,
* thanks NICT, KRISS, NTSC, PTB, OP, NIST, VNIIFTRI, INRIM, AOS, and NIM for
* their technical support and feedback, and
* thank BIPM and CCTF WG on TWSTFT for coordinating the interational activities.
*/

#include <stdio.h>
#include <math.h>
#include <hipfft/hipfft.h>
#include <stdlib.h>
#include <time.h>
#include <gsl/gsl_fit.h>
#include <pthread.h>
#include <hipblas.h>
#include "sample_v170215.hpp"

typedef struct _channel_info
{
  bool is_first; // 0x1 = first track; 0x0 = not first track
  bool is_trk;   // 0x1 = being tracked; 0x0 = no signal
  bool is_sic;   // 0x1 = successive interference cancellation (SIC); 0x0 = normal
  bool is_chA;   // 0x1 = physical ch A; 0x0 = ch B
  int prnno;     // PRN number
  int rc;        // number of chip per second (chips)
  int pt;        // coarse code head position (samples)
  int clen;      // number of chip per code length
  int *code;        // PRN code sequence {-1, 0, 1}
  int *cuda_code;   // PRN code sequence {-1, 0, 1} on GPU memory
  hipfftDoubleComplex *cuda_prn;      // freq domain code samples
  hipfftDoubleComplex *cuda_prn_t;    // time domain code waveform
  hipfftDoubleComplex *cuda_prn_acq;  // freq domain code waveform (ACQ)
  double fc_start; // central freq. from parameter file
  double fc;   // carrier freq. (Hz)
  double df;
  double gd;   // code phase (ns)
  double dg;   // code phase rate (ns/s)
  double phi;  // carrier phase (cycle)
  double last_phi; // last carrier phase (cycle)
  double peak; // peak value
  double prn_power; // power of the reference signal (V^2)
  double fmin; // lower (negative) cut-off freq (Hz) of the low-pass filter
  double fmax; // upper (positive) cut-off freq (Hz) of the low-pass filter
  double range; // acquisition frequency range (Hz)
  double step;  // acquisition frequency step (Hz)
  double snr_min; // minimum required SNR (dB)
  int n_stop;
  int n_start;
} channel_info;

typedef struct _fx_result
{
  FILE *fout;
  int count;        // available code arrival time measurements per second
  int count_sic;    // available code arrival time measurements, SIC, per second
  int *pidx;
  double *ttag_gd;  // time reference of the code arrival time
  double *gd;       // code arrival time (ns) on ttag_gd
  double *gd_sic;   // code arrival time, SIC (ns) on ttag_gd
  double *ttag_phi; // time reference of the carrier phase
  double *phi;      // carrier phase (cycle) on ttag_phi
  double *phi_raw;  // carrier phase (cycle) on the reference time
  double *w;        // indicator for code, 1 = available; 0 = no use
  double *w_sic;    // indicator for code, SIC, 1 = available; 0 = no use
  double *amp;      // signal amplitude (V)
  double *signal;   // signal power (V^2)
  double *noise;    // noise power (V^2)
} fx_result;

typedef struct _system_info
{
  int nch;      // number of channels
  int ntrk;     // number of channels in tracking state
  int nsic;     // number of channels for SIC
  int nobs;     // number of samples per code length
  int sps;      // samples per second
  int portion;  // number of codes per second
  double fs;    // sampling frequency (Hz)
  double period;// code length (s)
} system_info;

/* function prototype */
int CAcode(channel_info *);
int NICTcode(channel_info *);
int SATREcode(channel_info *);
double average(int, double *, double *);
double kth_smallest(double *, int, int);

void current_time(int, int *, int *, int *, int *, int *, int *, int *, int *);
void date2doy(int , int, int, int *);
void doy2date(int, int, int *, int *);
int date2mjd(int, int, int);
void mjd2date(int, int *, int *, int *);
int doy2mjd(int, int);
void mjd2doy(int, int *, int *);

__global__ void SIC(double *, int, int, int, int, double, hipfftDoubleComplex *, int, double, double);
__global__ void PRN_sampling(int, int *, hipfftDoubleComplex *, int, double, int, double);
__global__ void down_conversion(int, double, double, double *, hipfftDoubleComplex *);
__global__ void down_conversion2(int, double, double, char *, hipfftDoubleComplex *, bool);
__global__ void cross_spectrum(int, hipfftDoubleComplex *, hipfftDoubleComplex *, double, double, double, hipfftDoubleComplex *);
__global__ void binary_to_waveform(int, char *, double *, bool);
__global__ void conv(int, int, hipfftDoubleComplex *, double *, double *);

/* MAIN PROCESS */
int main(int argc, char *argv[])
{
  // check input arguments
  if (argc != 1 && argc != 7)
  {
    printf("use ./rx50 or ./rx50 yyyy MM dd hh mm ss\n");
    return 1;
  }

  FILE *fparam, *flog;
  samplebuffer sb;
  pthread_t tsampling;
  timetag *ttag;
  system_info sinfo; sinfo.sps = 50000000; sinfo.nch = 0;
  channel_info *cinfo;
  fx_result *fxres;
  hipfftHandle plan, plan_trk, plan_acq, plan_sic;
  hipfftDoubleComplex *cuda_xcor_fx, *cuda_robs, *cuda_robs_acq;

  hipSetDevice(0); // identify the GPU card for computation
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  char str[200], str2[200], result_dir[80], filename[80], paramfile[80], prn_type[10], ch[2], id[2], *pch;
  double c0, c1, chisq, peak, c00, c01, c11;
  double phi, x1, x2, x3, x4, x5, rx_power, *rpow;
  double fcc, fbest, frange, fstep, flow, fhigh;
  double global_peak, init_peak, *cuda_ddbs;
  int i, k, p, peak_idx, blocks, global_peak_idx, cnt;
  char *cuda_samples;
  double *cuda_xcor_conv, *xcor_conv, *cuda_xcor_phi, *xcor_phi;
  int ii, jj, idx, pn, rc, imax;
  double *res, tmp, snr_min, fc_start, ftrmax;
  bool is_chA, is_sic;
  struct tm *timeinfo;
  timeinfo = (struct tm *)malloc(sizeof(struct tm));
  
  // adjustable variables
  int nobs = 20; // maximum delay spread (+-samps)
  int nch_max = 8; // maximum number of software channels

  srand(time(NULL)); // random pick-up for ACQ
  sprintf(prn_type, "SATRE");
  sprintf(result_dir, "../result");
  sprintf(paramfile, "satre.param");

  // assign system parameters
  if ((fparam = fopen(paramfile, "r")) == NULL)
  {
    printf("no such parameter file : %s\n", paramfile);
    return 1;
  }
  fgets(str, 200, fparam); // discard the first raw

  // assign channel parameters and function result
  ttag = (timetag *)malloc(sizeof(timetag));
  cinfo = (channel_info *)malloc(sizeof(channel_info) * nch_max);
  fxres = (fx_result *)malloc(sizeof(fx_result) * nch_max);

  // modify parameters
  if (strcmp(prn_type, "GNSS") == 0)
    sinfo.period = 0.001; // 1ms
  else if (strcmp(prn_type, "NICT") == 0)
    sinfo.period = 0.002; // 2ms
  else if (strcmp(prn_type, "SATRE") == 0)
    sinfo.period = 0.004; // 4ms
  else
  {
    printf("PRN type error: %s", prn_type);
    return 1;
  }
  sinfo.nobs = (int)round((double)sinfo.sps * sinfo.period);
  sinfo.portion = (int)round(1.0 / sinfo.period);
  sinfo.fs = (double)sinfo.sps;
  hipfftPlan1d(&plan, sinfo.nobs, HIPFFT_Z2Z, 1);
  hipfftPlan1d(&plan_acq, sinfo.nobs * 2, HIPFFT_Z2Z, 1);

  //printf("sampling frequency : %.0lf Hz\n", sinfo.fs);
  //printf("code period : %lf sec\n", sinfo.period);
  //printf("nobs : %d\n", sinfo.nobs);
  //printf("portions : %d\n", sinfo.portion);

  // assign memories
  for (i = 0; i < nch_max; i++)
  {
    cinfo[i].prnno = 0;
    cinfo[i].fc_start = 0.0;
    cinfo[i].rc = 0;
    cinfo[i].fmax = 0.0;
    cinfo[i].fmin = 0.0;
    cinfo[i].is_chA = 0x0;
    cinfo[i].is_sic = 0x0;
    cinfo[i].range = 1.0;
    cinfo[i].step = 1.0;
    cinfo[i].snr_min = 0.0;
    cinfo[i].is_trk = 0x0;
    cinfo[i].is_first = 0x0;
    cinfo[i].fc = 0.0;
    cinfo[i].pt = 0;
    cinfo[i].gd = 0.0;
    cinfo[i].dg = 0.0;
    cinfo[i].phi = 0.0;
    cinfo[i].n_stop = 0;
    cinfo[i].n_start = 0;
    cinfo[i].clen = 0;
    cinfo[i].peak = 0.0;
    cinfo[i].last_phi = 0.0;
    hipMalloc((void **)&cinfo[i].cuda_prn, sizeof(hipfftDoubleComplex) * sinfo.nobs);
    hipMalloc((void **)&cinfo[i].cuda_prn_t, sizeof(hipfftDoubleComplex) * sinfo.nobs);
    hipMalloc((void **)&cinfo[i].cuda_prn_acq, sizeof(hipfftDoubleComplex) * sinfo.nobs * 2);
    fxres[i].pidx     = (int *)malloc(sizeof(int) * sinfo.portion);
    fxres[i].gd       = (double *)malloc(sizeof(double) * sinfo.portion);
    fxres[i].gd_sic   = (double *)malloc(sizeof(double) * sinfo.portion);
    fxres[i].ttag_gd  = (double *)malloc(sizeof(double) * sinfo.portion);
    fxres[i].phi      = (double *)malloc(sizeof(double) * sinfo.portion);
    fxres[i].ttag_phi = (double *)malloc(sizeof(double) * sinfo.portion);
    fxres[i].signal   = (double *)malloc(sizeof(double) * sinfo.portion);
    fxres[i].noise    = (double *)malloc(sizeof(double) * sinfo.portion);
    fxres[i].w        = (double *)malloc(sizeof(double) * sinfo.portion);
    fxres[i].w_sic    = (double *)malloc(sizeof(double) * sinfo.portion);
    fxres[i].amp      = (double *)malloc(sizeof(double) * sinfo.portion);
  }

  // initialize the buffer of samples
  for (i = 0; i < MAXBUF; i++)
  {
    sb.ttag[i] = (timetag *)malloc(sizeof(timetag));
    memset(sb.ttag[i], 0x0, sizeof(timetag));
    sb.buf[i] = (char *)malloc(sizeof(char) * sps * 2);
    if (sb.buf[i] == NULL)
    {
      printf("sample buffer error\n");
      return 1;
    }
    memset(sb.buf[i], 0, sizeof(char) * sps * 2);
  }

  // set time stamp
  if (argc == 1) sb.is_tt = 0x0;
  else sb.is_tt = 0x1;
  if (sb.is_tt == 0x1)
  {
    sscanf(argv[1], "%d", &timeinfo->tm_year);
    timeinfo->tm_year -= 1900;
    sscanf(argv[2], "%d", &timeinfo->tm_mon);
    timeinfo->tm_mon -= 1;
    sscanf(argv[3], "%d", &timeinfo->tm_mday);
    sscanf(argv[4], "%d", &timeinfo->tm_hour);
    sscanf(argv[5], "%d", &timeinfo->tm_min);
    sscanf(argv[6], "%d", &timeinfo->tm_sec);
    sb.tt = mktime(timeinfo);
  }
  free(timeinfo);
  sb.readd = -1;
  sb.writed = -1;

  hipMalloc((void **)&cuda_robs_acq, sizeof(hipfftDoubleComplex) * sinfo.nobs * 2);
  hipMalloc((void **)&cuda_samples, sizeof(char) * sinfo.sps * 2);

  rpow = (double *)malloc(sizeof(double) * nch_max);
  hipMalloc((void **)&cuda_robs, sizeof(hipfftDoubleComplex) * sinfo.nobs * nch_max);
  hipMalloc((void **)&cuda_xcor_fx, sizeof(hipfftDoubleComplex) * sinfo.nobs * nch_max);
  hipMalloc((void **)&cuda_ddbs, sizeof(double) * sinfo.nobs);

  hipMalloc((void **)&cuda_xcor_conv, sizeof(double) * (nobs * 2 + 1));
  hipMalloc((void **)&cuda_xcor_phi, sizeof(double) * (nobs * 2 + 1));
  xcor_conv = (double *)malloc(sizeof(double) * (nobs * 2 + 1));
  xcor_phi = (double *)malloc(sizeof(double) * (nobs * 2 + 1));
  res = (double *)malloc(sizeof(double) * sinfo.portion);

  // create a thread and start sampling
  pthread_mutex_init(&sb.mymutex, NULL);
  pthread_cond_init(&sb.mycond, NULL);
  pthread_create(&tsampling, NULL, sampling, (void *)&sb);

////////////////////////////// infinite observation //////////////////////////

  while (1)
  {
    // Get samples from buffer
    pthread_mutex_lock(&sb.mymutex);
    sb.readd++;
    if (sb.readd > sb.writed || sb.writed == -1)
      pthread_cond_wait(&sb.mycond, &sb.mymutex); // wait until the first sample
    hipMemcpy(cuda_samples, sb.buf[sb.readd % MAXBUF], sizeof(char) * sps * 2, hipMemcpyHostToDevice);
    memcpy(ttag, sb.ttag[sb.readd % MAXBUF], sizeof(timetag));
    pthread_mutex_unlock(&sb.mymutex);

    // Start: set new parameters and refresh every 30 seconds
    if (sinfo.nch == 0 || ttag->second == 17 || ttag->second == 47)
    {
      if ((fparam = fopen(paramfile, "r")) != NULL)
      {
        i = 0;
        while (fgets(str, 200, fparam))
        {
          if (str[0] == '#') continue;
          strcpy(str2, str);
          k = 0;
			    pch = (char *)strtok(str2, " ;\r\n");
			    if (pch != NULL) k++;
			    while ((pch = (char *)strtok(NULL, " ;\r\n")) != NULL) k++;
          if ((str[0] == 'A' || str[0] == 'B') && (str[2] == 'N' || str[2] == 'S') && k == 9)
          {
            if (str[0] == 'A') is_chA = 0x1; // physical channel A
            if (str[0] == 'B') is_chA = 0x0; // physical channel B
            if (str[2] == 'S') is_sic = 0x1; // with SIC
            if (str[2] == 'N') is_sic = 0x0; // normal
            sscanf(str, "%s %s %d %lf %d %lf %lf %lf %lf", ch, id, &pn, &fc_start, &rc, &ftrmax, &frange, &fstep, &snr_min);

            if (cinfo[i].is_chA == is_chA && cinfo[i].is_sic == is_sic && cinfo[i].prnno == pn && cinfo[i].fc_start == fc_start && cinfo[i].rc == rc * 1000 && cinfo[i].fmax == ftrmax * 1.0e+3 && cinfo[i].range >= frange && cinfo[i].range < frange * 2.0 && cinfo[i].step >= fstep && cinfo[i].step < fstep * 2.0 && cinfo[i].snr_min == snr_min)
            {
              i++;
              if (i < nch_max) continue;
              else break;
            }
            else if (pn >= 0 && pn <= 31 && fc_start > 1.0e+7 && fc_start < 1.0e+8 && (rc == 2500 || rc == 1000) && ftrmax > 0.0 && ftrmax < 1.0e+5 && frange > 0.0 && frange < 1.0e+6 && frange > fstep && snr_min > -100.0)
            {
              cinfo[i].prnno = pn;
              cinfo[i].fc_start = fc_start;
              cinfo[i].rc = rc * 1000;
              cinfo[i].fmax = ftrmax * 1.0e+3;
              cinfo[i].fmin = -cinfo[i].fmax;
              cinfo[i].is_chA = is_chA;
              cinfo[i].is_sic = is_sic;
              cinfo[i].range = 1.0;
              while (cinfo[i].range < frange) cinfo[i].range *= 2.0;
              cinfo[i].step = 1.0;
              while (cinfo[i].step < fstep) cinfo[i].step *= 2.0;
              cinfo[i].snr_min = snr_min;
              cinfo[i].is_trk = 0x0;
              cinfo[i].is_first = 0x1;
              cinfo[i].fc = 0.0;
              cinfo[i].pt = 0;
              cinfo[i].gd = 0.0;
              cinfo[i].dg = 0.0;
              cinfo[i].phi = 0.0;
              cinfo[i].n_stop = (int)floor(cinfo[i].fmax * sinfo.period);
              cinfo[i].n_start = (int)floor(cinfo[i].fmin * sinfo.period);
              cinfo[i].clen = 0;
              cinfo[i].peak = 0.0;
              cinfo[i].last_phi = 0.0;
    
              // obtain code length
              if (strcmp(prn_type, "GNSS") == 0) cinfo[i].clen = 1023; // GNSS
              else if (strcmp(prn_type, "NICT") == 0) cinfo[i].clen = 4095; // NICT modem
              else if (strcmp(prn_type, "SATRE") == 0)
              {
                if (cinfo[i].rc == 2500000) cinfo[i].clen = 10000; // SATRE modem
                else if (cinfo[i].rc == 1000000) cinfo[i].clen = 4000; // SATRE modem
                else
                {
                  printf("code rate error (new parameter): %d\n", cinfo[i].rc);
                  i++;
                  if (i < nch_max) continue;
                  else break;
                }
              }

              // assign memory for PRN code
              cinfo[i].code = (int *)malloc(sizeof(int) * cinfo[i].clen);
              hipMalloc((void **)&cinfo[i].cuda_code, sizeof(int) * cinfo[i].clen);
              if (strcmp(prn_type, "GNSS") == 0) CAcode(&cinfo[i]); // GNSS
              else if (strcmp(prn_type, "NICT") == 0) NICTcode(&cinfo[i]); // NICT modem
              else if (strcmp(prn_type, "SATRE") == 0) SATREcode(&cinfo[i]); // SATRE modem

              hipMemcpy(cinfo[i].cuda_code, cinfo[i].code, sizeof(int) * cinfo[i].clen, hipMemcpyHostToDevice);
              // assign GPU memory for PRN samples and spectrum
              hipMemset(cinfo[i].cuda_prn_acq, 0x0, sizeof(hipfftDoubleComplex) * sinfo.nobs * 2);
              blocks = sinfo.nobs / 1000;
              PRN_sampling<<<blocks, 1000>>>(sinfo.nobs, cinfo[i].cuda_code, cinfo[i].cuda_prn, cinfo[i].rc, sinfo.fs, cinfo[i].clen, 0.0); // cuda_prn = +- 1
              hipMemcpy(cinfo[i].cuda_prn_t, cinfo[i].cuda_prn, sizeof(hipfftDoubleComplex) * sinfo.nobs, hipMemcpyDeviceToDevice); // cuda_prn_t = +- 1
              hipMemcpy(cinfo[i].cuda_prn_acq, cinfo[i].cuda_prn, sizeof(hipfftDoubleComplex) * sinfo.nobs, hipMemcpyDeviceToDevice);
              hipfftExecZ2Z(plan, cinfo[i].cuda_prn, cinfo[i].cuda_prn, HIPFFT_FORWARD);
              hipfftExecZ2Z(plan_acq, cinfo[i].cuda_prn_acq, cinfo[i].cuda_prn_acq, HIPFFT_FORWARD);

              // compute reference signal power Pc (baseband filtered PRN)
              hipblasDznrm2(handle, abs(cinfo[i].n_start), (hipDoubleComplex *)cinfo[i].cuda_prn + sinfo.nobs - abs(cinfo[i].n_start), 1, &peak);
              hipblasDznrm2(handle, abs(cinfo[i].n_stop), (hipDoubleComplex *)cinfo[i].cuda_prn, 1, &rx_power);
              cinfo[i].prn_power = pow(peak / (double)sinfo.nobs, 2) + pow(rx_power / (double)sinfo.nobs, 2); // power in V^2

              //printf("#%02d, reference signal power: %lf dBm\n", cinfo[i].prnno, 10.0 * log10(cinfo[i].prn_power * 1000.0 / 50.0));

              // initialize memories
              memset(fxres[i].pidx    , 0x0, sizeof(int)    * sinfo.portion);
              memset(fxres[i].gd      , 0x0, sizeof(double) * sinfo.portion);
              memset(fxres[i].gd_sic  , 0x0, sizeof(double) * sinfo.portion);
              memset(fxres[i].ttag_gd , 0x0, sizeof(double) * sinfo.portion);
              memset(fxres[i].phi     , 0x0, sizeof(double) * sinfo.portion);
              memset(fxres[i].ttag_phi, 0x0, sizeof(double) * sinfo.portion);
              memset(fxres[i].signal  , 0x0, sizeof(double) * sinfo.portion);
              memset(fxres[i].noise   , 0x0, sizeof(double) * sinfo.portion);
              memset(fxres[i].w       , 0x0, sizeof(double) * sinfo.portion);
              memset(fxres[i].w_sic   , 0x0, sizeof(double) * sinfo.portion);
              memset(fxres[i].amp     , 0x0, sizeof(double) * sinfo.portion);

              // LOG
              sprintf(filename, "%s/%4d%02d%02d%02d.log", result_dir, ttag->year, ttag->month, ttag->day, ttag->hour);
              flog = fopen(filename, "a");
              fprintf(flog, "%02d:%02d:%02d + %11.9lf set param: #%02d %8.0lf %4d %5.0lf %5.0lf %5.0lf %3.0lf\n", ttag->hour, ttag->minute, ttag->second, ttag->fsec, cinfo[i].prnno, cinfo[i].fc_start, cinfo[i].rc / 1000, cinfo[i].fmax * 1.0e-3, cinfo[i].range, cinfo[i].step, cinfo[i].snr_min);
              fclose(flog);
              i++;
              if (i < nch_max) continue;
              else break;
            }
          }
        }
        fclose(fparam);
        sinfo.nch = i;
      }
    } // End: set new parameters and refresh every 30 seconds

    // Start: two-stage acquisition
    for (p = 0; p < 2; p++)
    {
      for (i = 0; i < sinfo.nch; i++)
      {
        if ((p == 0 && cinfo[i].is_trk == 0x0) || (p == 1 && cinfo[i].is_trk == 0x0 && cinfo[i].fc != 0.0))
        {
          if (p == 0) 
          {
            fbest = cinfo[i].fc_start;
            frange = cinfo[i].range;
            fstep = cinfo[i].step;
          }
          else
          {
            fbest = cinfo[i].fc;
            frange = cinfo[i].range / 2.0;
            fstep = cinfo[i].step / 2.0;
          }
          init_peak = 0.0;
          rx_power = 0.0;
          idx =  2 * (rand() % (sinfo.portion - 1)) * sinfo.nobs;
          while (1)
          {
            flow = fbest - frange;
            fhigh = fbest + frange;
            for (fcc = flow; fcc <= fhigh; fcc += fstep)
            {
              blocks = sinfo.nobs * 2 / 1000;
              // down convert
              down_conversion2<<<blocks, 1000>>>(sinfo.nobs * 2, fcc / sinfo.fs, 0.0, cuda_samples + idx, cuda_robs_acq, cinfo[i].is_chA);
              hipfftExecZ2Z(plan_acq, cuda_robs_acq, cuda_robs_acq, HIPFFT_FORWARD);

              // compute reception power (Px) in the last search period
              if (fstep >= 1.0 && fstep < 2.0 && rx_power == 0.0)
              {
                hipblasDznrm2(handle, abs(cinfo[i].n_start * 2), (hipDoubleComplex *)cuda_robs_acq + 2 * sinfo.nobs - abs(cinfo[i].n_start * 2), 1, &peak);
                hipblasDznrm2(handle, abs(cinfo[i].n_stop * 2) , (hipDoubleComplex *)cuda_robs_acq, 1, &rx_power);
                rx_power = pow(peak / 2.0 / (double)sinfo.nobs, 2) + pow(rx_power / 2.0 / (double)sinfo.nobs, 2); // reception power (V^2)
              }

              // perform cross correlation
              cross_spectrum<<<blocks, 1000>>>(sinfo.nobs * 2, cuda_robs_acq, cinfo[i].cuda_prn_acq, sinfo.fs / (double)sinfo.nobs / 2.0, cinfo[i].fmax, cinfo[i].fmin, cuda_robs_acq);
              hipfftExecZ2Z(plan_acq, cuda_robs_acq, cuda_robs_acq, HIPFFT_BACKWARD);

              // find peak
              hipblasIzamax(handle, sinfo.nobs, (hipDoubleComplex *)cuda_robs_acq, 1, &peak_idx);
              peak_idx -= 1; // hipblasIzamax() returns [1:nfft], but we want [0:nfft-1]
              hipblasDznrm2(handle, 1, (hipDoubleComplex *)cuda_robs_acq + peak_idx, 1, &peak);

              if (init_peak == 0.0)
              {
                init_peak = peak;
                global_peak = peak;
              }
              if (peak > global_peak)
              {
                global_peak = peak;
                fbest = fcc;
                global_peak_idx = peak_idx;
              }
            }
            // update to finer search range
            frange = fstep;
            fstep = fstep / 2.0;
            if (fstep < 1.0)
              break;
          }
          global_peak = 8.0 * global_peak * global_peak / cinfo[i].prn_power; // signal power (V^2)
          snr_min = pow(10.0, cinfo[i].snr_min / 10.0);
          if ((1.0 + snr_min) * global_peak > snr_min * rx_power) // say signal exists if SNR > required SNR
          {
            if (p == 0)
            {
              cinfo[i].fc = fbest;
              cinfo[i].gd = (double)global_peak_idx * 1.0e+9 / sinfo.fs;
              cinfo[i].pt = global_peak_idx;
              cinfo[i].peak = global_peak;
            }
            else
            {
              cinfo[i].fc = floor((fbest + cinfo[i].fc) / 2.0);
              cinfo[i].gd = (double)(global_peak_idx + cinfo[i].pt) * 1.0e+9 / sinfo.fs / 2.0;
              cinfo[i].pt = (global_peak_idx + cinfo[i].pt) / 2;
              cinfo[i].peak = (cinfo[i].peak + global_peak) / 2.0;
              cinfo[i].is_trk = 0x1;
            }
            // LOG
            sprintf(filename, "%s/%4d%02d%02d%02d.log", result_dir, ttag->year, ttag->month, ttag->day, ttag->hour);
            flog = fopen(filename, "a");
            if (p == 0) fprintf(flog, "%02d:%02d:%02d + %11.9lf ACQ1: PRN#%2d: %3d %8.0lf %7.0lf %6d %8.3lf %8.3lf\n", ttag->hour, ttag->minute, ttag->second, ttag->fsec, cinfo[i].prnno, idx / 2 / sinfo.nobs, cinfo[i].fc, cinfo[i].gd, cinfo[i].pt, 10.0 * log10(cinfo[i].peak * 1000.0 / 50.0), 10.0 * log10(rx_power * 1000.0 / 50.0));
            else
            {
              if (cinfo[i].is_chA == 0x1) printf("A: #%02d %4.1lf Mcps Code Lock\n", cinfo[i].prnno, (double)cinfo[i].rc * 1.0e-6);
              else printf("B: #%02d %4.1lf Mcps Code Lock\n", cinfo[i].prnno, (double)cinfo[i].rc * 1.0e-6);
              fprintf(flog, "%02d:%02d:%02d + %11.9lf ACQ2: PRN#%2d: %3d %8.0lf %7.0lf %6d %8.3lf %8.3lf\n", ttag->hour, ttag->minute, ttag->second, ttag->fsec, cinfo[i].prnno, idx / 2 / sinfo.nobs, cinfo[i].fc, cinfo[i].gd, cinfo[i].pt, 10.0 * log10(cinfo[i].peak * 1000.0 / 50.0), 10.0 * log10(rx_power * 1000.0 / 50.0));
            }
            fclose(flog);
            // LOG
          }
          else cinfo[i].fc = 0.0;
        }
      }
    } // End: two-stage acquisition

    sinfo.ntrk = 0;
    sinfo.nsic = 0;
    for (i = 0; i < sinfo.nch; i++)
    {
      if (cinfo[i].is_trk == 0x1)
      {
      	sinfo.ntrk++;
      	if (cinfo[i].is_sic == 0x1)
      	  sinfo.nsic++;
      }
    }
    if (sinfo.ntrk == 0) // if no desired PN code here, re-do ACQ for the next samples
    {
      printf("acquisition: %d , no signal\n", sinfo.nch);
      continue;
    }

    ttag->mjd = date2mjd(ttag->year, ttag->month, ttag->day);
    hipfftPlan1d(&plan_trk, sinfo.nobs, HIPFFT_Z2Z, sinfo.ntrk);
    hipfftPlan1d(&plan_sic, sinfo.nobs, HIPFFT_Z2Z, sinfo.nsic);
    for (i = 0; i < sinfo.nch; i++)
    {
      memset(fxres[i].w, 0x0, sizeof(double) * sinfo.portion);
      memset(fxres[i].w_sic, 0x0, sizeof(double) * sinfo.portion);
      fxres[i].count = 0;
      fxres[i].count_sic = 0;
    }

    // Start: Tracking
    for (p = 0; p < sinfo.portion - 1; p++)
    {
      // Start: measure code phase, amplitude, and dphi
      cnt = 0;
      blocks = sinfo.nobs / 1000;
      for (i = 0; i < sinfo.nch; i++)
      {
        if (cinfo[i].is_trk == 0x1)
        {
          phi = fmod(cinfo[i].fc * (double)(p * sinfo.nobs + cinfo[i].pt) / sinfo.fs, 1.0);
          idx = 2 * (p * sinfo.nobs + cinfo[i].pt);
          down_conversion2<<<blocks, 1000>>>(sinfo.nobs, cinfo[i].fc / sinfo.fs, phi, cuda_samples + idx, cuda_robs + cnt * sinfo.nobs, cinfo[i].is_chA);
          cnt++;
        }
      }
      hipfftExecZ2Z(plan_trk, cuda_robs, cuda_robs, HIPFFT_FORWARD);

      // compute reception power (Px)
      cnt = 0;
      for (i = 0; i < sinfo.nch; i++)
      {
        if (cinfo[i].is_trk == 0x1)
        {
          hipblasDznrm2(handle, abs(cinfo[i].n_start), (hipDoubleComplex *)cuda_robs + cnt * sinfo.nobs + sinfo.nobs - abs(cinfo[i].n_start), 1, &peak);
          hipblasDznrm2(handle, abs(cinfo[i].n_stop), (hipDoubleComplex *)cuda_robs + cnt * sinfo.nobs, 1, &rx_power);
          rpow[i] = pow(rx_power / (double)sinfo.nobs, 2) +  pow(peak / (double)sinfo.nobs, 2);
          cnt++;
        }
      }

      // perform cross correlation
      cnt = 0;
      for (i = 0; i < sinfo.nch; i++)
      {
        if (cinfo[i].is_trk == 0x1)
        {
          cross_spectrum<<<blocks, 1000>>>(sinfo.nobs, cuda_robs + cnt * sinfo.nobs, cinfo[i].cuda_prn, sinfo.fs / (double)sinfo.nobs, cinfo[i].fmax, cinfo[i].fmin, cuda_xcor_fx + cnt * sinfo.nobs);
          cnt++;
        }
      }
      hipfftExecZ2Z(plan_trk, cuda_xcor_fx, cuda_xcor_fx, HIPFFT_BACKWARD);

      cnt = 0;
      for (i = 0; i < sinfo.nch; i++)
      {
        if (cinfo[i].is_trk == 0x1)
        {
          fxres[i].ttag_gd[p] = (double)p * sinfo.period;
          fxres[i].ttag_phi[p] = (double)p * sinfo.period + (double)cinfo[i].pt / sinfo.fs;
          conv<<<1, 1000>>>(sinfo.nobs, nobs, cuda_xcor_fx + cnt * sinfo.nobs, cuda_xcor_conv, cuda_xcor_phi);
          hipblasIdamax(handle, nobs * 2 + 1, cuda_xcor_conv, 1, &peak_idx); // peak
          peak_idx -= 1;
          fxres[i].pidx[p] = peak_idx - nobs;
          if (peak_idx - nobs >= 0)
            hipblasDznrm2(handle, 1, (hipDoubleComplex *)cuda_xcor_fx + peak_idx - nobs + cnt * sinfo.nobs, 1, &peak);
          else
            hipblasDznrm2(handle, 1, (hipDoubleComplex *)cuda_xcor_fx + peak_idx - nobs + (cnt + 1) * sinfo.nobs, 1, &peak);

          // compute SNR
          // peak = A * Pc
          // amplitude: A (in V) = peak / Pc
          // reception power: Px (in V^2) = A^2 * Pc + Pn = rx_power
          // reference signal power: Pc (in V^2) = cinfo[i].prn_power
          // signal power: Ps (in V^2) = A^2 * Pc = peak * peak / Pc = fxres[i].signal
          // noise power: Pn (in V^2) = Px - Ps = fxres[i].noise
          // SNR = A^2 * Pc / Pn = 1.0 / ((Px * Pc / peak) - 1.0)
          fxres[i].amp[p] = peak / cinfo[i].prn_power; // amplitude, used for SIC and code head decision
          fxres[i].signal[p] = peak * peak / cinfo[i].prn_power; // signal power (in V^2)
          if (rpow[i] > fxres[i].signal[p]) fxres[i].noise[p] = rpow[i] - fxres[i].signal[p]; // noise power (in V^2)
          else fxres[i].noise[p] = 5.0e-7; // if Pn > Px, then noise power can be ignored (assigned by a small value)

          // proceed if peak_idx is within the range of the delay spread, and SNR meets min SNR requirement
          snr_min = pow(10.0, cinfo[i].snr_min / 10.0);
          if (peak_idx - 2 >= 0 && peak_idx + 2 < nobs * 2 + 1 && (1.0 + snr_min) * fxres[i].signal[p] > snr_min * rpow[i])
          {
            hipMemcpy(xcor_conv, cuda_xcor_conv, sizeof(double) * (nobs * 2 + 1), hipMemcpyDeviceToHost);
            hipMemcpy(xcor_phi, cuda_xcor_phi, sizeof(double) * (nobs * 2 + 1), hipMemcpyDeviceToHost);

            // compute the carrier phase
            fxres[i].phi[p] = xcor_phi[peak_idx];

            // compute the code phase
            x1 = xcor_conv[peak_idx - 2]; x2 = xcor_conv[peak_idx - 1]; x3 = xcor_conv[peak_idx];
            x4 = xcor_conv[peak_idx + 1]; x5 = xcor_conv[peak_idx + 2];
            // perform Narrow Correlator
            // fxres[i].gd[p] = ((x2 - x4) / (x2 - 2.0 * x3 + x4) / 2.0 + (double)(cinfo[i].pt + peak_idx - nobs)) * 1.0e+9 / sinfo.fs;
            // perform High Resolution Correlator
            fxres[i].gd[p] = ((x2 - x4) / (x2 - 2.0 * x3 + x4) - (x1 - x5) / (x1 - 2.0 * x3 + x5) + (double)(cinfo[i].pt + peak_idx - nobs)) * 1.0e+9 / sinfo.fs;
            if (fxres[i].gd[p] > sinfo.period * 1.0e+9) fxres[i].gd[p] -= (sinfo.period * 1.0e+9);
            else if (fxres[i].gd[p] < 0.0) fxres[i].gd[p] += (sinfo.period * 1.0e+9);
            fxres[i].w[p] = 1.0;
            fxres[i].count++;
          }
          cnt++;
        }
      } // End: measure code phase, amplitude, and dphi
      // Start: Successive Inteference Cancellation (SIC) for the (p - 1)th obs
      blocks = sinfo.nobs / 500;
      if (p > 1)
      {
        // use local phase and aquired frequency to down convert, and then accumulate
        cnt = 0;
        for (i = 0; i < sinfo.nch; i++)
        {
          if (cinfo[i].is_sic == 0x1 && cinfo[i].is_trk == 0x1) // the i-th desired signal
          {
            if (cinfo[i].is_chA == 0x1) idx = 2 * ((p - 1) * sinfo.nobs + cinfo[i].pt);
            else idx = 2 * ((p - 1) * sinfo.nobs + cinfo[i].pt);
            blocks = sinfo.nobs / 1000;
            binary_to_waveform<<<blocks, 1000>>>(sinfo.nobs, cuda_samples + idx, cuda_ddbs, cinfo[i].is_chA);
            for (k = 0; k < sinfo.nch; k++)
            {
              if (i != k && cinfo[i].is_chA == cinfo[k].is_chA && cinfo[k].is_trk) // the k-th interference
              {
                if (cinfo[k].pt > cinfo[i].pt) // desired signal comes earlier than the interference, use p - 1 & p - 2
                {
                  if (fxres[k].w[p - 2] != 0.0) // cancel head, p - 2
                    SIC<<<blocks, 1000>>>(cuda_ddbs, sinfo.nobs, 0, sinfo.nobs + cinfo[i].pt - cinfo[k].pt, sinfo.nobs - 1               , fxres[k].amp[p - 2], cinfo[k].cuda_prn_t, fxres[k].pidx[p - 2], (cinfo[k].fc + cinfo[k].df) / sinfo.fs, fxres[k].phi[p - 2] + cinfo[k].fc * (double)((p - 2) * sinfo.nobs + cinfo[k].pt) / sinfo.fs - cinfo[k].df * sinfo.period);
                  if (fxres[k].w[p - 1] != 0.0) // cancel tail, p - 1
                    SIC<<<blocks, 1000>>>(cuda_ddbs, sinfo.nobs, cinfo[k].pt - cinfo[i].pt, 0, sinfo.nobs + cinfo[i].pt - cinfo[k].pt - 1, fxres[k].amp[p - 1], cinfo[k].cuda_prn_t, fxres[k].pidx[p - 1], (cinfo[k].fc + cinfo[k].df) / sinfo.fs, fxres[k].phi[p - 1] + cinfo[k].fc * (double)((p - 1) * sinfo.nobs + cinfo[k].pt) / sinfo.fs - cinfo[k].df * sinfo.period);
                }
                else // desired signal comes later than the interference, use p - 1 & p
                {
                  if (fxres[k].w[p - 1] != 0.0) // cancel head, p - 1
                    SIC<<<blocks, 1000>>>(cuda_ddbs, sinfo.nobs, 0, cinfo[i].pt - cinfo[k].pt             , sinfo.nobs - 1               , fxres[k].amp[p - 1], cinfo[k].cuda_prn_t, fxres[k].pidx[p - 1], (cinfo[k].fc + cinfo[k].df) / sinfo.fs, fxres[k].phi[p - 1] + cinfo[k].fc * (double)((p - 1) * sinfo.nobs + cinfo[k].pt) / sinfo.fs - cinfo[k].df * sinfo.period);
                  if (fxres[k].w[p] != 0.0) // cancel tail, p
                    SIC<<<blocks, 1000>>>(cuda_ddbs, sinfo.nobs, sinfo.nobs + cinfo[k].pt - cinfo[i].pt, 0, cinfo[i].pt - cinfo[k].pt - 1, fxres[k].amp[p]    , cinfo[k].cuda_prn_t, fxres[k].pidx[p]    , (cinfo[k].fc + cinfo[k].df) / sinfo.fs, fxres[k].phi[p]     + cinfo[k].fc * (double)(p       * sinfo.nobs + cinfo[k].pt) / sinfo.fs - cinfo[k].df * sinfo.period);
                }
              }
            }
            phi = fmod(cinfo[i].fc * (double)((p - 1) * sinfo.nobs + cinfo[i].pt) / sinfo.fs, 1.0);
            down_conversion<<<blocks, 1000>>>(sinfo.nobs, cinfo[i].fc / sinfo.fs, phi, cuda_ddbs, cuda_robs + cnt * sinfo.nobs);
            cnt++;
          }
        }

        // perform cross correlation
        hipfftExecZ2Z(plan_sic, cuda_robs, cuda_robs, HIPFFT_FORWARD);
        cnt = 0;
        for (i = 0; i < sinfo.nch; i++)
        {
          if (cinfo[i].is_sic == 0x1 && cinfo[i].is_trk == 0x1)
          {
            cross_spectrum<<<blocks, 1000>>>(sinfo.nobs, cuda_robs + cnt * sinfo.nobs, cinfo[i].cuda_prn, sinfo.fs / (double)sinfo.nobs, cinfo[i].fmax, cinfo[i].fmin, cuda_xcor_fx + cnt * sinfo.nobs);
            cnt++;
          }
        }
        hipfftExecZ2Z(plan_sic, cuda_xcor_fx, cuda_xcor_fx, HIPFFT_BACKWARD);

        cnt = 0;
        for (i = 0; i < sinfo.nch; i++)
        {
          if(cinfo[i].is_sic == 0x1 && cinfo[i].is_trk == 0x1)
          {
            conv<<<1, 1000>>>(sinfo.nobs, nobs, cuda_xcor_fx + cnt * sinfo.nobs, cuda_xcor_conv, cuda_xcor_phi);
            hipblasIdamax(handle, nobs * 2 + 1, cuda_xcor_conv, 1, &peak_idx); // peak
			      peak_idx -= 1;
            hipMemcpy(xcor_conv, cuda_xcor_conv, sizeof(double) * (nobs * 2 + 1), hipMemcpyDeviceToHost);
            hipMemcpy(xcor_phi, cuda_xcor_phi, sizeof(double) * (nobs * 2 + 1), hipMemcpyDeviceToHost);
            if (peak_idx - 2 >= 0 && peak_idx + 2 < nobs * 2 + 1) // proceed if peak_idx is in the range of the delay spread
            {
              // compute the code phase
              fxres[i].ttag_gd[p - 1] = (double)(p - 1) * sinfo.period;
              x1 = xcor_conv[peak_idx - 2]; x2 = xcor_conv[peak_idx - 1]; x3 = xcor_conv[peak_idx];
              x4 = xcor_conv[peak_idx + 1]; x5 = xcor_conv[peak_idx + 2];
              // perform Narrow Correlator
              // fxres[i].gd_sic[p - 1] = ((x2 - x4) / (x2 - 2.0 * x3 + x4) / 2.0 + (double)(cinfo[i].pt + peak_idx - nobs)) * 1.0e+9 / sinfo.fs;
              // perform High Resolution Correlator
              fxres[i].gd_sic[p - 1] = ((x2 - x4) / (x2 - 2.0 * x3 + x4) - (x1 - x5) / (x1 - 2.0 * x3 + x5) + (double)(cinfo[i].pt + peak_idx - nobs)) * 1.0e+9 / sinfo.fs;
              // remove delay code phase ambiguity
              if (fxres[i].gd_sic[p - 1] > sinfo.period * 1.0e+9) fxres[i].gd_sic[p - 1] -= (sinfo.period * 1.0e+9);
              else if (fxres[i].gd_sic[p - 1] < 0.0) fxres[i].gd_sic[p - 1] += (sinfo.period * 1.0e+9);
              fxres[i].w_sic[p - 1] = 1.0;
              fxres[i].count_sic++;
            }
            cnt++;
          }
        }
      } // End: SIC
    } // End: Tracking

    hipfftDestroy(plan_trk);
    hipfftDestroy(plan_sic);

    if (sinfo.ntrk != 0) printf("acquisition: %d , lock: %d \n", sinfo.nch - sinfo.ntrk, sinfo.ntrk);
    for (i = 0; i < sinfo.nch; i++)
    {
      if (cinfo[i].is_trk == 0x1)
      {
        if (fxres[i].count_sic * 2 > sinfo.portion && cinfo[i].is_sic == 0x1 && cinfo[i].is_first == 0x0) // SIC result
        {
          // Start: update and record
          if (cinfo[i].is_chA == 0x1) sprintf(filename, "%s/%4d%02d%02d%02d.chA.pn%02d.%04dkcps.sic.dat", result_dir, ttag->year, ttag->month, ttag->day, ttag->hour, cinfo[i].prnno, cinfo[i].rc / 1000);
          else sprintf(filename, "%s/%4d%02d%02d%02d.chB.pn%02d.%04dkcps.sic.dat", result_dir, ttag->year, ttag->month, ttag->day, ttag->hour, cinfo[i].prnno, cinfo[i].rc / 1000);
          fxres[i].fout = fopen(filename, "a");
          fprintf(fxres[i].fout, "%2d %2d %2d ", ttag->hour, ttag->minute, ttag->second);

          // filter out the obs larger than 3 sigma
          ii = 0;
          memset(res, 0x0, sizeof(double) * sinfo.portion);
          for (p = 0; p < sinfo.portion; p++)
          {
            if (fxres[i].w_sic[p] > 0.0)
            {
              res[ii] = fxres[i].gd_sic[p];
              ii++;
            }
          }
          c0 = kth_smallest(res, ii, ii / 2);
          chisq = (kth_smallest(res, ii, ii * 3 / 4) - kth_smallest(res, ii, ii / 4)) / 1.349;
          fxres[i].count_sic = 0;
          for (p = 0; p < sinfo.portion; p++)
          {
            if (fxres[i].w_sic[p] != 0.0)
              if (fabs(fxres[i].gd_sic[p] - c0) > 3.0 * chisq)
                fxres[i].w_sic[p] = 0.0;
            if (fxres[i].w_sic[p] > 0.0) fxres[i].count_sic++;
          }

          // code phase
          gsl_fit_wlinear(fxres[i].ttag_gd, 1, fxres[i].w_sic, 1, fxres[i].gd_sic, 1, sinfo.portion, &c0, &c1, &c00, &c01, &c11, &chisq);
          chisq = sqrt(chisq / (double)fxres[i].count_sic);
          fprintf(fxres[i].fout, "%5d %3d %14.6lf %14.6lf %8.3lf\n", ttag->hour * 3600 + ttag->minute * 60 + ttag->second, fxres[i].count_sic, c0 + 0.5 * c1, c1, chisq);
          fclose(fxres[i].fout); // close output file

          // End: update and record
          // reset all the measurements
          memset(fxres[i].gd_sic , 0, sizeof(double) * sinfo.portion);
        }

        // Normal result
        if (fxres[i].count * 2 > sinfo.portion)
        {
          // find head of the code sequence 
          imax = 0;
          tmp = 0.0;
          if (cinfo[i].is_first == 0x0)
          {
            memset(res, 0x0, sizeof(double) * sinfo.portion);
            if (cinfo[i].rc == 1000000) // for 1Mcps SATRE code
            {
              fxres[i].amp[sinfo.portion - 1] = (fxres[i].amp[0] + fxres[i].amp[sinfo.portion - 2]) / 2.0;
              for (ii = 0; ii < sinfo.portion; ii++)
              {
                for (jj = 0; jj < 16; jj++)
                {
                  idx = (ii + jj) % sinfo.portion;
                  res[ii] += fxres[i].amp[idx];
                }
              }
              tmp = res[0];
              for (ii = 1; ii < sinfo.portion; ii++)
              {
                if (res[ii] < tmp)
                {
                  imax = ii;
                  tmp = res[ii];
                }
              }
              for (ii = -1; ii < 17; ii++) // -1 ~ 16
              {
                jj = (ii + imax + sinfo.portion) % sinfo.portion;
                fxres[i].w[jj] = 0.0;
              }
            }
            else if (cinfo[i].rc == 2500000) // for 2.5Mcps SATRE code
            {
              tmp = kth_smallest(fxres[i].gd, sinfo.portion, sinfo.portion / 2); // find the median among the obs
              for (ii = 0; ii < sinfo.portion; ii++) // fill the median
              {
                if (fxres[i].w[ii] == 0.0)
                  fxres[i].gd[ii] = tmp;
              }
              for (ii = 0; ii < sinfo.portion - 1; ii++)
                res[ii] = fxres[i].gd[ii] - fxres[i].gd[ii + 1];
              res[sinfo.portion - 1] = fxres[i].gd[sinfo.portion - 1] - fxres[i].gd[0];
              tmp = res[0];
              for (ii = 1; ii < sinfo.portion; ii++)
              {
                if (res[ii] > tmp)
                {
                  imax = ii;
                  tmp = res[ii];
                }
              }
              fxres[i].w[imax] = 1.0;
              fxres[i].gd[imax] -= 200.0;
              fxres[i].w[(imax + 1) % sinfo.portion] = 1.0;
              fxres[i].gd[(imax + 1) % sinfo.portion] += 200.0;
            }
          }

          // filter out the obs larger than 3 sigma
          ii = 0;
          memset(res, 0x0, sizeof(double) * sinfo.portion);
          for (p = 0; p < sinfo.portion; p++)
          {
            if (fxres[i].w[p] > 0.0)
            {
              res[ii] = fxres[i].gd[p];
              ii++;
            }
          }
          c0 = kth_smallest(res, ii, ii / 2);
          chisq = (kth_smallest(res, ii, ii * 3 / 4) - kth_smallest(res, ii, ii / 4)) / 1.349;
          fxres[i].count = 0;
          for (p = 0; p < sinfo.portion; p++)
          {
            if (fxres[i].w[p] != 0.0)
            {
              if (fabs(fxres[i].gd[p] - c0) < 3.0 * chisq)
              {
                fxres[i].count++;
                while (fabs(fxres[i].phi[p] - cinfo[i].last_phi) > 0.25) // BPSK phase adjustment
                {
                  if (fxres[i].phi[p] > cinfo[i].last_phi) fxres[i].phi[p] -= 0.5;
                  else fxres[i].phi[p] += 0.5;
                }
                cinfo[i].last_phi = fxres[i].phi[p];
              }
              else fxres[i].w[p] = 0.0;
            }
          }

          // Start: update and record
          if (cinfo[i].is_first == 0x0)
          {
            if (cinfo[i].is_chA == 0x1) sprintf(filename, "%s/%4d%02d%02d%02d.chA.pn%02d.%04dkcps.dat", result_dir, ttag->year, ttag->month, ttag->day, ttag->hour, cinfo[i].prnno, cinfo[i].rc / 1000);
            else sprintf(filename, "%s/%4d%02d%02d%02d.chB.pn%02d.%04dkcps.dat", result_dir, ttag->year, ttag->month, ttag->day, ttag->hour, cinfo[i].prnno, cinfo[i].rc / 1000);
            fxres[i].fout = fopen(filename, "a");
            fprintf(fxres[i].fout, "%2d %2d %2d ", ttag->hour, ttag->minute, ttag->second);
          }

          // apply linear fit, and then update fc and pt
          gsl_fit_wlinear(fxres[i].ttag_phi, 1, fxres[i].w, 1, fxres[i].phi, 1, sinfo.portion, &c0, &c1, &c00, &c01, &c11, &chisq);
          cinfo[i].fc += floor(c1);
          cinfo[i].df = c1 - floor(c1);
          cinfo[i].phi = fmod(c0 + 1000.0, 1.0);
          if (cinfo[i].is_chA == 0x1) printf("A: #%02d %4.1lf Mcps %12.3lf Hz ", cinfo[i].prnno, (double)cinfo[i].rc * 1.0e-6, cinfo[i].fc + cinfo[i].df);
          else printf("B: #%02d %4.1lf Mcps %12.3lf Hz ", cinfo[i].prnno, (double)cinfo[i].rc * 1.0e-6, cinfo[i].fc + cinfo[i].df);
          if (cinfo[i].is_first == 0x0) fprintf(fxres[i].fout, "%14.6lf %11.8lf ", cinfo[i].fc + c1 - floor(c1), cinfo[i].phi);

          // code phase
          gsl_fit_wlinear(fxres[i].ttag_gd, 1, fxres[i].w, 1, fxres[i].gd, 1, sinfo.portion, &c0, &c1, &c00, &c01, &c11, &chisq);
          chisq = sqrt(chisq / (double)fxres[i].count);
          cinfo[i].gd = c0 + 0.5 * c1;
          cinfo[i].dg = c1;
          cinfo[i].pt = (int)round((c0 + c1) * sinfo.fs / 1.0e+9);
          printf("%13.3lf (%5.3lf) ns ", cinfo[i].gd + (double)imax * sinfo.period * 1.0e+9, chisq / sqrt((double)fxres[i].count));
          if (cinfo[i].is_first == 0x0) fprintf(fxres[i].fout, "%5d %3d %5.3lf %14.6lf %11.6lf %8.3lf ", ttag->hour * 3600 + ttag->minute * 60 + ttag->second, fxres[i].count, (double)imax * sinfo.period, cinfo[i].gd, cinfo[i].dg, chisq);

          // signal power in dBm
          cinfo[i].peak = average(sinfo.portion, fxres[i].signal, fxres[i].w);
          if (cinfo[i].is_first == 0x0) fprintf(fxres[i].fout, "%7.3lf ", 10.0 * log10(cinfo[i].peak * 1000.0 / 50.0));

          // noise power in dBm
          peak = average(sinfo.portion, fxres[i].noise, fxres[i].w);
          printf("S/N %6.2lf dB\n", 10.0 * log10(cinfo[i].peak / peak)); // show S/N in screen
          if (cinfo[i].is_first == 0x0) fprintf(fxres[i].fout, "%7.3lf\n", 10.0 * log10(peak * 1000.0 / 50.0));

          if (cinfo[i].is_first == 0x0) fclose(fxres[i].fout); // close output file

          // End: update and record
          // reset all the measurements
          memset(fxres[i].phi    , 0, sizeof(double) * sinfo.portion);
          memset(fxres[i].gd     , 0, sizeof(double) * sinfo.portion);
          memset(fxres[i].noise  , 0, sizeof(double) * sinfo.portion);
          memset(fxres[i].signal , 0, sizeof(double) * sinfo.portion);
          cinfo[i].is_first = 0x0;
        }
        else
        {
          // LOG
          sprintf(filename, "%s/%4d%02d%02d%02d.log", result_dir, ttag->year, ttag->month, ttag->day, ttag->hour);
          flog = fopen(filename, "a");
          if (cinfo[i].is_chA == 0x1)
          {
            //printf("Ch. A, PRN#%2d, count = %d / %d ,loss of lock\n", cinfo[i].prnno, fxres[i].count, sinfo.portion);
            fprintf(flog, "%02d:%02d:%02d + %11.9lf loss of lock: Ch. A, PRN#%2d, count = %d / %d\n", ttag->hour, ttag->minute, ttag->second, ttag->fsec, cinfo[i].prnno, fxres[i].count, sinfo.portion);
          }
          else
          {
            //printf("Ch. B, PRN#%2d, count = %d / %d ,loss of lock\n", cinfo[i].prnno, fxres[i].count, sinfo.portion);
            fprintf(flog, "%02d:%02d:%02d + %11.9lf loss of lock: Ch. B, PRN#%2d, count = %d / %d\n", ttag->hour, ttag->minute, ttag->second, ttag->fsec, cinfo[i].prnno, fxres[i].count, sinfo.portion);
          }
          fclose(flog);
          // LOG
          cinfo[i].fc = 0.0;
          cinfo[i].is_first = 0x1;
          cinfo[i].is_trk = 0x0;
          cinfo[i].dg = 0.0;
          cinfo[i].phi = 0.0;
          cinfo[i].last_phi = 0.0;
          // reset all the measurements
          memset(fxres[i].phi    , 0, sizeof(double) * sinfo.portion);
          memset(fxres[i].gd     , 0, sizeof(double) * sinfo.portion);
          memset(fxres[i].noise  , 0, sizeof(double) * sinfo.portion);
          memset(fxres[i].signal , 0, sizeof(double) * sinfo.portion);
          //continue;
        }
      }
    }
  }
  return 0;
}

double kth_smallest(double *a, int n, int k) // find the k-th smallest value in the array a[]
{
  int i, j, l, m;
  double x, *b, t;
  b = (double *)malloc(sizeof(double) * n);
  memcpy(b, a, sizeof(double) * n);
  l = 0; m = n - 1;
  while (l < m)
  {
    x = b[k]; i = l; j = m;
    do
    {
      while (b[i] < x) i++;
      while (b[j] > x) j--;
      if (i <= j)
      {
        t = b[i]; b[i] = b[j]; b[j] = t;
        i++; j--;
      }
    } while (i <= j);
    if (j < k) l = i;
    if (k < i) m = j;
  }
  x = b[k];
  free(b);
  return x;
}

int SATREcode(channel_info *c)
{
  int i, j, shift_back, reg[14];
  int tap[32][14] = {{  1,	1,  0,	0,  0,	0,  1,	0,  1,	0,  0,	0,  0,	0}, /* PN # 0 */
                     {  1,	1,  0,	0,  0,	0,  1,	0,  0,	0,  0,	1,  0,	0}, /* PN # 1 */
                     {  1,	0,  1,	0,  1,	0,  0,	0,  0,	0,  0,	0,  0,	1}, /* PN # 2 */
                     {  1,	0,  0,	0,  0,	0,  0,	0,  0,	1,  0,	1,  0,	1}, /* PN # 3 */
                     {  1,	0,  0,	0,  0,	0,  1,	0,  0,	0,  0,	1,  1,	0}, /* PN # 4 */
                     {  1,	1,  0,	0,  0,	0,  0,	0,  0,	0,  0,	1,  1,	0}, /* PN # 5 */
                     {  1,	1,  0,	0,  0,	0,  1,	0,  0,	0,  1,	0,  0,	0}, /* PN # 6 */
                     {  1,	1,  1,	0,  0,	0,  0,	0,  0,	0,  0,	0,  1,	0}, /* PN # 7 */
                     {  1,	0,  0,	1,  0,	0,  1,	1,  0,	0,  0,	0,  1,	1}, /* PN # 8 */
                     {  1,	1,  0,	0,  0,	0,  0,	0,  0,	1,  0,	1,  1,	1}, /* PN # 9 */
                     {  1,	0,  0,	1,  0,	0,  0,	0,  1,	1,  0,	0,  1,	1}, /* PN #10 */
                     {  1,	0,  0,	1,  0,	0,  1,	1,  0,	0,  0,	0,  1,	1}, /* PN #11 */
                     {  1,	0,  0,	0,  1,	0,  0,	0,  1,	1,  1,	0,  0,	1}, /* PN #12 */
                     {  1,	0,  1,	1,  0,	0,  0,	1,  0,	1,  0,	0,  0,	1}, /* PN #13 */
                     {  1,	0,  1,	0,  0,	0,  0,	0,  1,	0,  1,	1,  1,	0}, /* PN #14 */
                     {  1,	0,  0,	0,  0,	1,  0,	1,  1,	0,  0,	1,  1,	0}, /* PN #15 */
                     {  1,  0,  0,  0,  0,  1,  1,  1,  0,  0,  1,  0,  1,  0}, /* PN #16 */
                     {  1,  0,  0,  1,  1,  1,  0,  0,  0,  0,  1,  0,  1,  0}, /* PN #17 */
                     {  1,  0,  0,  1,  0,  1,  0,  0,  1,  1,  0,  0,  1,  0}, /* PN #18 */
                     {  1,  0,  0,  0,  1,  0,  1,  1,  1,  0,  0,  0,  1,  0}, /* PN #19 */
                     {  1,  1,  0,  0,  1,  0,  1,  0,  1,  0,  0,  0,  1,  0}, /* PN #20 */
                     {  1,  0,  1,  1,  1,  0,  1,  0,  0,  0,  0,  0,  1,  0}, /* PN #21 */
                     {  1,  0,  0,  0,  1,  0,  1,  0,  0,  1,  1,  1,  0,  0}, /* PN #22 */
                     {  1,  0,  1,  0,  1,  0,  1,  0,  0,  0,  1,  1,  0,  0}, /* PN #23 */
                     {  1,  1,  0,  0,  0,  1,  0,  1,  0,  1,  0,  1,  0,  0}, /* PN #24 */
                     {  1,  0,  0,  1,  0,  1,  1,  0,  1,  0,  0,  1,  0,  0}, /* PN #25 */
                     {  1,  0,  0,  0,  1,  1,  0,  0,  1,  1,  1,  0,  0,  0}, /* PN #26 */
                     {  1,  0,  1,  0,  1,  1,  0,  0,  0,  1,  1,  0,  0,  0}, /* PN #27 */
                     {  1,  1,  0,  0,  0,  1,  1,  1,  0,  0,  1,  0,  0,  0}, /* PN #28 */
                     {  1,  1,  1,  0,  0,  0,  0,  1,  1,  1,  0,  0,  0,  0}, /* PN #29 */
                     {  1,  1,  1,  0,  0,  1,  0,  1,  1,  0,  0,  0,  0,  0}, /* PN #30 */
                     {  1,  1,  1,  1,  1,  0,  1,  0,  0,  0,  0,  0,  0,  0}};/* PN #31 */
  if (c->prnno < 0 || c->prnno > 31)
  {
    printf("no such prn number, %d\n", c->prnno);
    return 1;
  }

  for (i = 0; i < 14; i++)
    reg[i] = 1;
  for (i = 0; i < c->clen; i++)
  {
    if (c->clen == 4000 && i < 10)
      c->code[i] = 0; // 0
    else
      c->code[i] = 1 - 2 * reg[0]; // -1, 1
    shift_back = reg[0];
    for (j = 1; j < 14; j++)
      if (tap[c->prnno][j] == 1)
        shift_back ^= reg[j];
    for (j = 1; j < 14; j++)
      reg[j - 1] = reg[j];
    reg[13] = shift_back;
  }
  return 0;
}

int NICTcode(channel_info *c)
{
  int i, j, shift_back, reg[12];
  int tap[8][12] = {{ 1,	0,  0,	0,  0,	0,  1,	0,  1,	0,  0,	1},	 /* PN #0 */
                    { 1,	1,  0,	1,  1,	1,  0,	1,  0,	0,  1,	1},	 /* PN #1 */
                    { 1,	0,  0,	1,  0,	0,  0,	0,  0,	1,  1,	0},	 /* PN #2 */
                    { 1,	0,  1,	1,  1,	0,  1,	1,  1,	0,  1,	0},	 /* PN #3 */
                    { 1,	1,  0,	0,  0,	1,  0,	1,  1,	1,  0,	0},	 /* PN #4 */
                    { 1,	0,  0,	1,  1,	1,  0,	1,  0,	0,  0,	1},	 /* PN #5 */
                    { 1,	1,  1,	0,  0,	1,  0,	1,  1,	1,  0,	1},	 /* PN #6 */
                    { 1,	0,  1,	0,  1,	1,  1,	0,  1,	1,  1,	0}}; /* PN #7 */
  if (c->prnno < 0 || c->prnno > 7)
  {
    printf("no such prn number, %d\n", c->prnno);
    return 1;
  }

  for (i = 0; i < 12; i++)
    reg[i] = 1;
  for (i = 0; i < 4095; i++)
  {
    shift_back = reg[0];
    for (j = 1; j < 12; j++)
      if (tap[c->prnno][j] == 1) shift_back ^= reg[j];

    c->code[i] = shift_back; // 0, 1

    for (j = 1; j < 12; j++)
      reg[j - 1] = reg[j];
    reg[11] = shift_back;
  }
  return 0;
}

/* GNSS Coarse Aquisition code */
int CAcode(channel_info *c)
{
  int i, j, res, shift_back;
  int code_phase[37][2] = {{ 2,  6}, { 3,  7}, { 4,  8}, { 5,  9}, { 1,  9},
                           { 2, 10}, { 1,  8}, { 2,  9}, { 3, 10}, { 2,  3},
                           { 3,  4}, { 5,  6}, { 6,  7}, { 7,  8}, { 8,  9},
                           { 9, 10}, { 1,  4}, { 2,  5}, { 3,  6}, { 4,  7},
                           { 5,  8}, { 6,  9}, { 1,  3}, { 4,  6}, { 5,  7},
                           { 6,  8}, { 7,  9}, { 8, 10}, { 1,  6}, { 2,  7},
                           { 3,  8}, { 4,  9}, { 5, 10}, { 4, 10}, { 1,  7},
                           { 2,  8}, { 4, 10}};
  int *G1_reg, *G2_reg;
  if (c->prnno < 1 || c->prnno > 37)
  {
    printf("no such prn number, %d\n", c->prnno);
    return 1;
  }
  G1_reg = (int *)malloc(sizeof(int) * 10);
  G2_reg = (int *)malloc(sizeof(int) * 10);
  for (i = 0; i < 10; i++)
  {
    G1_reg[i] = 1;
    G2_reg[i] = 1;
  }
  for (i = 0; i < 1023; i++)
  {
    res = G1_reg[9] ^ G2_reg[code_phase[c->prnno - 1][0] - 1] ^ G2_reg[code_phase[c->prnno - 1][1] - 1];
    c->code[i] = res; // 0, 1

    /* G1 */
    shift_back = G1_reg[2] ^ G1_reg[9];
    for (j = 9; j >= 1; j--)
      G1_reg[j] = G1_reg[j - 1];
    G1_reg[0] = shift_back;

    /* G2 */
    shift_back = G2_reg[1] ^ G2_reg[2] ^ G2_reg[5] ^ G2_reg[7] ^ G2_reg[8] ^ G2_reg[9];
    for (j = 9; j >= 1; j--)
      G2_reg[j] = G2_reg[j - 1];
    G2_reg[0] = shift_back;
  }
  free(G1_reg);
  free(G2_reg);
  return 0;
}

double average(int nobs, double *x, double *w)
{
	int i, count = 0;
	double res = 0.0;
	if (nobs == 0)
	  return 0.0;
	for (i = 0; i < nobs; i++)
	{
		if (w[i] > 0.0)
		{
			res += x[i];
			count++;
		}
	}
	return res / (double)count;
}

__global__ void SIC(double *obs, int nobs, int start_pt, int init_pt, int end_pt, double amp, hipfftDoubleComplex *prn, int pidx, double ff, double phi)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= start_pt && i <= start_pt + end_pt - init_pt)
    obs[i] -= amp * prn[i - start_pt + init_pt].x * cos(2.0 * PI * (ff * (double)(i - start_pt + init_pt) + phi));
}

__global__ void PRN_sampling(int nobs, int *code, hipfftDoubleComplex *prn, int rc, double fs, int clen, double delay)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x, idx;
  if (i < nobs)
  {
    idx = (int)floor(fmod(((double)i / fs - delay * 1.0e-9) * (double)rc, (double)clen));
    if (idx < 0)
      idx += clen;
    else if (idx >= clen)
      idx -= clen;
    prn[i].x = (double)code[idx];
    prn[i].y = 0.0;
  }
}

__global__ void cross_spectrum(int nobs, hipfftDoubleComplex *obs, hipfftDoubleComplex *prn, double df, double fmax, double fmin, hipfftDoubleComplex *robs)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x, idx;
  if (i < nobs)
  {
    idx = (i >= nobs / 2) ? i - nobs : i;
    if ((double)idx * df < fmax && (double)idx * df > fmin && idx != 0)
    {
      robs[i].x = (obs[i].x * prn[i].x + obs[i].y * prn[i].y) / (double)nobs / (double)nobs;
      robs[i].y = (obs[i].y * prn[i].x - obs[i].x * prn[i].y) / (double)nobs / (double)nobs;
    }
    else
    {
      robs[i].x = 0.0;
      robs[i].y = 0.0;
    }
  }
}

/* down convertion, cuda_obs -> cuda_dobs
 * ff : digital frequency
 * phi : initial phase
 */
__global__ void down_conversion(int nobs, double ff, double phi, double *obs, hipfftDoubleComplex *dobs)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < nobs)
  {
    dobs[i].x = 1.4142135624 * obs[i] * cos(-1.0 * (2.0 * PI * ff * (double)i + phi * 2.0 * PI));
    dobs[i].y = 1.4142135624 * obs[i] * sin(-1.0 * (2.0 * PI * ff * (double)i + phi * 2.0 * PI));
  }
}

__global__ void down_conversion2(int nobs, double ff, double phi, char *samp, hipfftDoubleComplex *dobs, bool is_chA)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < nobs)
  {
    if (is_chA == 0x1)
    {
      dobs[i].x = 1.4142135624 * (double)samp[i * 2] * cos(-1.0 * (2.0 * PI * ff * (double)i + phi * 2.0 * PI)) / 128.0;
      dobs[i].y = 1.4142135624 * (double)samp[i * 2] * sin(-1.0 * (2.0 * PI * ff * (double)i + phi * 2.0 * PI)) / 128.0;
    }
    else
    {
      dobs[i].x = 1.4142135624 * (double)samp[i * 2 + 1] * cos(-1.0 * (2.0 * PI * ff * (double)i + phi * 2.0 * PI)) / 128.0;
      dobs[i].y = 1.4142135624 * (double)samp[i * 2 + 1] * sin(-1.0 * (2.0 * PI * ff * (double)i + phi * 2.0 * PI)) / 128.0;
    }
  }
}

__global__ void binary_to_waveform(int nobs, char *samp, double *obs, bool is_chA)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < nobs)
  {
    if (is_chA == 0x1) obs[i] = (double)samp[2 * i] / 128.0;
    else obs[i] = (double)samp[2 * i + 1] / 128.0;
  }
}

__global__ void conv(int snobs, int nobs, hipfftDoubleComplex *in, double *out, double *phi) // Cartesian to polar coordinate conversion
{
  int i = blockIdx.x * blockDim.x + threadIdx.x, idx;
  if (i <= 2 * nobs)
  {
    idx = (i - nobs < 0) ? i - nobs + snobs : i - nobs;
    out[i] = (in[idx].x * in[idx].x + in[idx].y * in[idx].y);
    phi[i] = atan2(in[idx].y, in[idx].x) / 2.0 / PI; // phase in cycle
  }
}

/*
	current time (UTC + 0)
*/
void current_time(int offset, int *mjd, int *year, int *month, int *day, int *doy, int *hour, int *minute, int *second)
{
	time_t sec;
	sec = (int)time(NULL) + offset;
	*mjd = 40587 + sec / 86400;
	*hour = (sec % 86400) / 3600;
	*minute = (sec % 3600) / 60;
	*second = sec % 60;
	mjd2doy(*mjd, year, doy);
	mjd2date(*mjd, year, month, day);
//	return 40587.0 + (double)time(NULL) / 86400.0;
	return;
}

void date2doy(int year, int month, int day, int *doy)
{

	int i, mday[13] = {0, 31, 28, 31, 30, 31, 30, 31, 31, 30, 31, 30, 31};
	if ((year % 4 == 0 && year % 100 != 0 ) || year % 400 == 0)
		mday[2] = 29;
  *doy = day;
  for (i = 1; i < month; i++)
  	*doy += mday[i];
	return;
}

void doy2date(int year, int doy, int *month, int *day)
{
	int mday[13] = {0, 31, 28, 31, 30, 31, 30, 31, 31, 30, 31, 30, 31};
	if ((year % 4 == 0 && year % 100 != 0 ) || year % 400 == 0)
		mday[2] = 29;
	*month = 1;
	*day = doy;
	while (*day > mday[*month])
  {
    *day -= mday[*month];
    (*month)++;
  }
	return;
}

/*
  date to MJD
*/
int date2mjd(int year, int month, int day)
{
  int mday[13] = {0, 31, 28, 31, 30, 31, 30, 31, 31, 30, 31, 30, 31};
  int i, leap_days;
  leap_days = (year - 1) / 4 - (year - 1) / 100 + (year - 1) / 400;
  if ((year % 4 == 0 && year % 100 != 0 ) || year % 400 == 0)
    mday[2] = 29;
  for (i = 1; i < month; i++)
    day = day + mday[i];
  return (year - 1) * 365 + day + leap_days - 678576;
}
/*
  MJD to date
*/
void mjd2date(int mjd, int *year, int *month, int *day)
{
  int mday [13] = {0, 31, 28, 31, 30, 31, 30, 31, 31, 30, 31, 30, 31};
  // 400 years = 146097 days, 100 years = 36524 days, 4 years = 1461 days, 3 normal years = 1095
  *year = 1; *month = 1;
  *day = mjd + 678576; //total days from 0001/01/01
  *year += *day / 146097 * 400; *day = *day % 146097;
  *year += *day /  36524 * 100; *day = *day %  36524;
  *year += *day /   1461 *   4; *day = *day %   1461;

  if (*day <= 1095)
  {
  	*year += *day / 365;
  	*day = *day % 365;
  }
  else
  {
  	*year += 3;
  	*day -= 1095;
  }

  if (*day == 0)
  {
  	(*year)--;
  	*month = 12;
  	*day = 31;
  	return;
  }

  if (( *year % 4 == 0 && *year % 100 != 0 ) || *year % 400 == 0)
  	mday[2] = 29;


  while (*day > mday[*month])
  {
    *day -= mday[*month];
    (*month)++;
  }
  return;
}
/*
  DOY (day of year) to MJD
*/
int doy2mjd(int year, int doy)
{
  int leap_days;
  leap_days = (year - 1) / 4 - (year - 1) / 100 + (year - 1) / 400;
  return (year - 1) * 365 + doy + leap_days - 678576;
}
/*
  MJD to DOY (day of year)
*/
void mjd2doy(int mjd, int *year, int *doy)
{
	// 400 years = 146097 days, 100 years = 36524 days, 4 years = 1461 days
	*year = 1;
	*doy = mjd + 678576;
	*year += *doy / 146097 * 400; *doy = *doy % 146097;
	*year += *doy /  36524 * 100; *doy = *doy %  36524;
	*year += *doy /   1461 *   4; *doy = *doy %   1461;

	if (*doy <= 1095)
  {
  	*year += *doy / 365;
  	*doy = *doy % 365;
  }
  else
  {
  	*year += 3;
  	*doy -= 1095;
  }

	if (*doy == 0)
	{
		(*year)--;
		if (( *year % 4 == 0 && *year % 100 != 0 ) || *year % 400 == 0)
			*doy = 366;
		else
			*doy = 365;
	}

	return;
}
